#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include "Imagen.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// t actual size of the image: 1024, 512, 256
// s es el ancho de la imagen inicial
__global__ void
HWT_1D_h( double * e, double * sal, int s, int t )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int dz = blockIdx.z * (s*s) ;
	//si excedimos la mitad horizontal o todas las filas
	if( x >= ( t + 1) / 2 || y >= t ) return;
	
	const double sqrt2 = 2;//1.414213;
	//s es el ancho de la imagen
	int i = ( y * s + 2 * x ) + dz ;//el *2 es sporque se avanza de dos en dos: 0,2,4,6, debo agarrar la x-ésima pareja de números
	int ii = ( y * s + x) + dz;//posición exacta dentro de la matriz gral
	
	sal[ii] = (e[i] + e[i+1])/sqrt2;
	sal[ii+t/2] = (e[i] - e[i+1])/sqrt2;

}


__global__ void
HWT_1D_v ( double * e, double * sal, int s, int t)
{
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int dz = blockIdx.z * (s*s) ;

	if(y >= (t+1)/2 || x >= t ) return;

	int o = (2 *  y * s + x) + dz ;
	int p = o + s;
	int q = y*s+x + dz;

	const double sqrt2 = 2;// 1.4142135;
	
	sal[q] = (e[o] + e[p]) / sqrt2;
	sal[q+s*t/2] = (e[o] - e[p])/sqrt2;
}


//t = tamaño total n*n*3
void HWT_2D(double* h_imagen , int t, int n){
	double* d_e;
	double* d_s;

	//tamaño total  en bytes de la imagen
	int tam_i = t * sizeof(double);
	

	hipMalloc(&d_e, tam_i);
	hipMalloc(&d_s, tam_i);

	
	hipMemcpy(d_e, h_imagen , tam_i, hipMemcpyHostToDevice);

	dim3 bloque(32,32);//1024 maxThreadsPerBlock
	dim3 grid;

	t /= 3;//tamano de cada matriz
	t = sqrt(t);//ancho de la imagen

	grid.x = ( t + bloque.x - 1) / bloque.x;//~~32
	grid.y =(t + bloque.y - 1) / bloque.y;//~~32
	grid.z = 3;

	grid.x *= 0.5;//solo se necesitan n/2 hilos

	int i = t;
	
	float tiempo;
	hipEvent_t inicio, fin;
	hipEventCreate(&inicio);
	hipEventCreate(&fin);

	hipEventRecord(inicio, 0);

	while(i > 1 && n > 0 ){
		HWT_1D_h<<<grid,bloque>>>(d_e, d_s, t, i);
		hipDeviceSynchronize();
		HWT_1D_v<<<grid, bloque>>>(d_s, d_e, t, i);
		hipDeviceSynchronize();
		i /= 2;
		n--;
	}

	hipEventRecord(fin, 0);
	hipEventSynchronize(fin);
	hipEventElapsedTime(&tiempo, inicio, fin);

	printf("Tiempo de transformación paralelo: %3.2f ms \n", tiempo);
	
	hipMemcpy(h_imagen, d_e, tam_i, hipMemcpyDeviceToHost);
	
	hipFree(d_e); hipFree(d_s);
}

 

int main(int argc, char **argv){
    if(argc<4) { printf("Uso: ./ejecutable Npasos entrada salida\n");   exit(0);}

    double * matriz;
    double * vectorBGR;
    int n = atoi(argv[1]);
    int t;
    char *entrada = argv[2];	
    char *salida = argv[3];	
    Imagen * imagen;

    imagen = new Imagen();

    //obtener matriz de imagen
    matriz = imagen->leerBMP(entrada);

    //obtener tamano de la imagen
    t = imagen->getTamano();
    
    vectorBGR = imagen->separarRGB(matriz);

    HWT_2D(vectorBGR, t, n);

    matriz = imagen->unirRGB(vectorBGR);

    imagen->guardarBMP(salida, matriz);
    return 0;
}
